#include "hip/hip_runtime.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

struct mse_functor
{
  mse_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = x-y;
      return z*z;
  }
};


static int cunn_MSECriterion_forward(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, torch_CudaTensor_id);
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, torch_CudaTensor_id);
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");

  float sum;

  long size = THCudaTensor_nElement(input);

  input = THCudaTensor_newContiguous(input);
  target = THCudaTensor_newContiguous(target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), mse_functor());

  if(sizeAverage)
    sum /= size;

  THCudaTensor_free(input);
  THCudaTensor_free(target);
 
  lua_pushnumber(L, sum);
  lua_setfield(L, 1, "output");

  lua_pushnumber(L, sum);
  return 1;
}


struct mse_backward_functor
{
  const float norm;

  mse_backward_functor(float norm_) : norm(norm_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      return norm * (x - y);
  }
};

static int cunn_MSECriterion_backward(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, torch_CudaTensor_id);
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, torch_CudaTensor_id);
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", torch_CudaTensor_id);

  long size = THCudaTensor_nElement(input);
  float norm = (sizeAverage ? 2./size : 2.);

  input = THCudaTensor_newContiguous(input);
  target = THCudaTensor_newContiguous(target);

  THCudaTensor_resizeAs(gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, mse_backward_functor(norm));

  THCudaTensor_free(input);
  THCudaTensor_free(target);
  return 1;
}

static const struct luaL_Reg cunn_MSECriterion__ [] = {
  {"MSECriterion_forward", cunn_MSECriterion_forward},
  {"MSECriterion_backward", cunn_MSECriterion_backward},
  {NULL, NULL}
};

static void cunn_MSECriterion_init(lua_State *L)
{
  luaT_pushmetaclass(L, torch_CudaTensor_id);
  luaT_registeratname(L, cunn_MSECriterion__, "nn");
  lua_pop(L,1);
}
